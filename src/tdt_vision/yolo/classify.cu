#include "hip/hip_runtime.h"
#include "infer.hpp"
#include "classify.hpp"

namespace classify{
    using namespace std;
    #define GPU_BLOCK_THREADS 512
    #define checkRuntime(call)                                              \
        do {                                                                 \
            auto ___call__ret_code__ = (call);                                \
            if (___call__ret_code__ != hipSuccess) {                          \
            INFO("CUDA Runtime error💥 %s # %s, code = %s [ %d ]", #call,       \
                hipGetErrorString(___call__ret_code__),                         \
                hipGetErrorName(___call__ret_code__), ___call__ret_code__);      \
            abort();                                                               \
            }                                                                       \
        } while (0)
    inline int upbound(int n, int align = 32) {
        return (n + align - 1) / align * align;
    }

    #define checkKernel(...)                 \
    do {                                      \
        { (__VA_ARGS__); }                     \
        checkRuntime(hipPeekAtLastError());    \
    } while (0)

    // static dim3 grid_dims(int numJobs) {
    //     int numBlockThreads =
    //         numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
    //     return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
    // }

    // static dim3 block_dims(int numJobs) {
    //     return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
    // }

    struct AffineMatrix {
    float i2d[6];  // image to dst(network), 2x3 matrix
    float d2i[6];  // dst to image, 2x3 matrix

    void compute(const std::tuple<int, int> &from,
                const std::tuple<int, int> &to) {
        float scale_x = get<0>(to) / (float)get<0>(from);
        float scale_y = get<1>(to) / (float)get<1>(from);
        float scale = std::min(scale_x, scale_y);
        i2d[0] = scale;
        i2d[1] = 0;
        i2d[2] = -scale * get<0>(from) * 0.5 + get<0>(to) * 0.5 + scale * 0.5 - 0.5;
        i2d[3] = 0;
        i2d[4] = scale;
        i2d[5] = -scale * get<1>(from) * 0.5 + get<1>(to) * 0.5 + scale * 0.5 - 0.5;

        double D = i2d[0] * i2d[4] - i2d[1] * i2d[3];
        D = D != 0. ? double(1.) / D : double(0.);
        double A11 = i2d[4] * D, A22 = i2d[0] * D, A12 = -i2d[1] * D,
            A21 = -i2d[3] * D;
        double b1 = -A11 * i2d[2] - A12 * i2d[5];
        double b2 = -A21 * i2d[2] - A22 * i2d[5];

        d2i[0] = A11;
        d2i[1] = A12;
        d2i[2] = b1;
        d2i[3] = A21;
        d2i[4] = A22;
        d2i[5] = b2;
    }
    };
    
    enum class NormType : int { None = 0, MeanStd = 1, AlphaBeta = 2 };

    enum class ChannelType : int { None = 0, SwapRB = 1 };

    int postprocess(vector<float> &output_array) {
        int max_index = 0;
        float max_value = output_array[0];
        for (int i = 1; i < output_array.size(); i++) {
            if (output_array[i] > max_value) {
                max_value = output_array[i];
                max_index = i;
            }
        }
        return max_index;
    }

    struct Norm {
        float mean[3];
        float std[3];
        float alpha, beta;
        NormType type = NormType::None;
        ChannelType channel_type = ChannelType::None;

        // out = (x * alpha - mean) / std
        static Norm mean_std(const float mean[3], const float std[3],
                            float alpha = 1 / 255.0f,
                            ChannelType channel_type = ChannelType::None);

        // out = x * alpha + beta
        static Norm alpha_beta(float alpha, float beta = 0,
                                ChannelType channel_type = ChannelType::None);

        // None
        static Norm None();
    };

    static __global__ void warp_affine_bilinear_and_normalize_plane_kernel(
    uint8_t *src, int src_line_size, int src_width, int src_height, float *dst,
    int dst_width, int dst_height, uint8_t const_value_st,
    float *warp_affine_matrix_2_3, Norm norm) {
        int dx = blockDim.x * blockIdx.x + threadIdx.x;
        int dy = blockDim.y * blockIdx.y + threadIdx.y;
        if (dx >= dst_width || dy >= dst_height) return;

        float m_x1 = warp_affine_matrix_2_3[0];
        float m_y1 = warp_affine_matrix_2_3[1];
        float m_z1 = warp_affine_matrix_2_3[2];
        float m_x2 = warp_affine_matrix_2_3[3];
        float m_y2 = warp_affine_matrix_2_3[4];
        float m_z2 = warp_affine_matrix_2_3[5];

        float src_x = m_x1 * dx + m_y1 * dy + m_z1;
        float src_y = m_x2 * dx + m_y2 * dy + m_z2;
        float c0, c1, c2;

        if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
            // out of range
            c0 = const_value_st;
            c1 = const_value_st;
            c2 = const_value_st;
        } else {
            int y_low = floorf(src_y);
            int x_low = floorf(src_x);
            int y_high = y_low + 1;
            int x_high = x_low + 1;

            uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
            float ly = src_y - y_low;
            float lx = src_x - x_low;
            float hy = 1 - ly;
            float hx = 1 - lx;
            float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
            uint8_t *v1 = const_value;
            uint8_t *v2 = const_value;
            uint8_t *v3 = const_value;
            uint8_t *v4 = const_value;
            if (y_low >= 0) {
            if (x_low >= 0) v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width) v2 = src + y_low * src_line_size + x_high * 3;
            }

            if (y_high < src_height) {
            if (x_low >= 0) v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width) v4 = src + y_high * src_line_size + x_high * 3;
            }

            // same to opencv
            c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
            c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
            c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
        }

        if (norm.channel_type == ChannelType::SwapRB) {
            float t = c2;
            c2 = c0;
            c0 = t;
        }

        if (norm.type == NormType::MeanStd) {
            c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
            c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
            c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
        } else if (norm.type == NormType::AlphaBeta) {
            c0 = c0 * norm.alpha + norm.beta;
            c1 = c1 * norm.alpha + norm.beta;
            c2 = c2 * norm.alpha + norm.beta;
        }

        int area = dst_width * dst_height;
        float *pdst_c0 = dst + dy * dst_width + dx;
        float *pdst_c1 = pdst_c0 + area;
        float *pdst_c2 = pdst_c1 + area;
        *pdst_c0 = c0;
        *pdst_c1 = c1;
        *pdst_c2 = c2;
    }

    static void warp_affine_bilinear_and_normalize_plane(
        uint8_t *src, int src_line_size, int src_width, int src_height, float *dst,
        int dst_width, int dst_height, float *matrix_2_3, uint8_t const_value,
        const Norm &norm, hipStream_t stream) {
    dim3 grid((dst_width + 31) / 32, (dst_height + 31) / 32);
    dim3 block(32, 32);

    checkKernel(warp_affine_bilinear_and_normalize_plane_kernel<<<grid, block, 0,
                                                                    stream>>>(
        src, src_line_size, src_width, src_height, dst, dst_width, dst_height,
        const_value, matrix_2_3, norm));
    }
    Norm Norm::mean_std(const float mean[3], const float std[3], float alpha,
                        ChannelType channel_type) {
    Norm out;
    out.type = NormType::MeanStd;
    out.alpha = alpha;
    out.channel_type = channel_type;
    memcpy(out.mean, mean, sizeof(out.mean));
    memcpy(out.std, std, sizeof(out.std));
    return out;
    }

    Norm Norm::alpha_beta(float alpha, float beta, ChannelType channel_type) {
    Norm out;
    out.type = NormType::AlphaBeta;
    out.alpha = alpha;
    out.beta = beta;
    out.channel_type = channel_type;
    return out;
    }

    Norm Norm::None() { return Norm(); }


    class InferImpl : public Infer {
    public:
    shared_ptr<trt::Infer> trt_;
    string engine_file_;
    Type type_;
    vector<shared_ptr<trt::Memory<unsigned char>>> preprocess_buffers_;
    trt::Memory<float> input_buffer_, output_array_;
    int num_class_=0;
    int network_input_width_, network_input_height_;
    bool isdynamic_model_ = false;
    Norm normalize_;
    float mean_[3], std_[3];
    // vector<int> bbox_head_dims_;
    // bool has_segment_ = false;
    // bool has_keyPoint = false;
    // bool isdynamic_model_ = false;
    // vector<shared_ptr<trt::Memory<unsigned char>>> box_segment_cache_;

    virtual ~InferImpl() = default;

    void adjust_memory(int batch_size) {//complete
        // the inference batch_size
        size_t input_numel = network_input_width_ * network_input_height_ * 3;
        input_buffer_.gpu(batch_size * input_numel);
        output_array_.gpu(batch_size * num_class_);
        output_array_.cpu(batch_size * num_class_);

        if ((int)preprocess_buffers_.size() < batch_size) {
        for (int i = preprocess_buffers_.size(); i < batch_size; ++i)
            preprocess_buffers_.push_back(
                make_shared<trt::Memory<unsigned char>>());
        }
    }

    void preprocess(int ibatch, const Image &image,
                    shared_ptr<trt::Memory<unsigned char>> preprocess_buffer,
                    AffineMatrix &affine, void *stream = nullptr) {
        affine.compute(make_tuple(image.width, image.height),
                    make_tuple(network_input_width_, network_input_height_));

        size_t input_numel = network_input_width_ * network_input_height_ * 3;
        float *input_device = input_buffer_.gpu() + ibatch * input_numel;
        size_t size_image = image.width * image.height * 3;
        size_t size_matrix = upbound(sizeof(affine.d2i), 32);
        uint8_t *gpu_workspace = preprocess_buffer->gpu(size_matrix + size_image);
        float *affine_matrix_device = (float *)gpu_workspace;
        uint8_t *image_device = gpu_workspace + size_matrix;

        uint8_t *cpu_workspace = preprocess_buffer->cpu(size_matrix + size_image);
        float *affine_matrix_host = (float *)cpu_workspace;
        uint8_t *image_host = cpu_workspace + size_matrix;

        // speed up
        hipStream_t stream_ = (hipStream_t)stream;
        std::chrono ::high_resolution_clock::time_point a1 =
            std::chrono::high_resolution_clock::now();
        memcpy(image_host, image.bgrptr, size_image);
        memcpy(affine_matrix_host, affine.d2i, sizeof(affine.d2i));
        std::chrono ::high_resolution_clock::time_point a2 =
            std::chrono::high_resolution_clock::now();
        auto time_used2 =
            std::chrono::duration_cast<std::chrono::duration<double>>(a2 - a1);
    //    INFO("memcpy time: %f", time_used2.count() * 1000);
        checkRuntime(hipMemcpyAsync(image_device, image_host, size_image,
                                    hipMemcpyHostToDevice, stream_));
        checkRuntime(hipMemcpyAsync(affine_matrix_device, affine_matrix_host,
                                    sizeof(affine.d2i), hipMemcpyHostToDevice,
                                    stream_));
        warp_affine_bilinear_and_normalize_plane(
            image_device, image.width * 3, image.width, image.height, input_device,
            network_input_width_, network_input_height_, affine_matrix_device, 114,
            normalize_, stream_);
    }

    bool load(const string &engine_file, Type type) {//complete
        trt_ = trt::load(engine_file);
        if (trt_ == nullptr) return false;

        trt_->print();

        this->type_ = type;

        auto input_dim = trt_->static_dims(0);
        auto output_dims_ = trt_->static_dims(1);
        puts("input_dim");
        for(int i=0;i<4;i++){
            std::cout<<input_dim[i]<<std::endl;
        }
        // puts(input_dim[1]);
        // puts(input_dim[2]);
        // puts(input_dim[3]);
        network_input_width_ = input_dim[3];
        network_input_height_ = input_dim[2];
        isdynamic_model_ = trt_->has_dynamic_dim();

        mean_[0] = 0.485;
        mean_[1] = 0.456;
        mean_[2] = 0.406;
        std_[0] = 0.229;
        std_[1] = 0.224;
        std_[2] = 0.225;
        normalize_ = Norm::mean_std(mean_, std_, 1.0 / 255.0, ChannelType::SwapRB);
        num_class_ = output_dims_[1];
        // std::cout<<"dims:";
        // std::cout<<output_dims_[0]<<std::endl;
        // std::cout<<output_dims_[1]<<std::endl;
        // std::cout<<output_dims_[2]<<std::endl;


        // if (type == Type::densenet121){

        // } else {
        // INFO("Unsupport type %d", type);
        // }
        return true;
    }

    virtual int forward(const Image &image,
                            void *stream = nullptr) override {
        auto output = forwards({image}, stream);
        if (output.empty()) return {};
        return output[0];
    }

    virtual vector<int> forwards(const vector<Image> &images,
                                        void *stream = nullptr) override {
        int num_image = images.size();
        if (num_image == 0) return {};

        auto input_dims = trt_->static_dims(0);
        int infer_batch_size = input_dims[0];
        if (infer_batch_size != num_image) {
        if (isdynamic_model_) {
            infer_batch_size = num_image;
            input_dims[0] = num_image;
            if (!trt_->set_run_dims(0, input_dims)) return {};
        } else {
            if (infer_batch_size < num_image) {
            INFO(
                "When using static shape model, number of images[%d] must be "
                "less than or equal to the maximum batch[%d].",
                num_image, infer_batch_size);
            return {};
            }
        }
        }

 std::chrono ::high_resolution_clock::time_point a1 =
        std::chrono::high_resolution_clock::now();
    adjust_memory(infer_batch_size);//调用内存
    std::chrono ::high_resolution_clock::time_point a2 =
        std::chrono::high_resolution_clock::now();
    auto time_used2 =
        std::chrono::duration_cast<std::chrono::duration<double>>(a2 - a1);
//    INFO("adjust_memory time: %f", time_used2.count() * 1000);

    vector<AffineMatrix> affine_matrixs(num_image);
    std::chrono ::high_resolution_clock::time_point a3 =
        std::chrono::high_resolution_clock::now();
    hipStream_t stream_ = (hipStream_t)stream;
    for (int i = 0; i < num_image; ++i)
      preprocess(i, images[i], preprocess_buffers_[i], affine_matrixs[i],
                 stream);

    vector<void *> bindings{input_buffer_.gpu(), output_array_.gpu()};
    //binding是什么

    std::chrono ::high_resolution_clock::time_point a3d1 =
        std::chrono::high_resolution_clock::now();
    auto time_used3 =
        std::chrono::duration_cast<std::chrono::duration<double>>(a3d1 - a3);
//    INFO("preprocess time: %f", time_used3.count() * 1000);
    if (!trt_->forward(bindings, stream)) {
      INFO("Failed to tensorRT forward.");
      return {};
    }
    std::chrono ::high_resolution_clock::time_point a3d2 =
        std::chrono::high_resolution_clock::now();
    auto time_used3d2 =
        std::chrono::duration_cast<std::chrono::duration<double>>(a3d2 - a3d1);
//    INFO("trt forward time: %f", time_used3d2.count() * 1000);

    std::chrono ::high_resolution_clock::time_point a3d3 =
        std::chrono::high_resolution_clock::now();
    auto time_used3d3 =
        std::chrono::duration_cast<std::chrono::duration<double>>(a3d3 - a3d2);
//    INFO("decode_kernel_invoker time: %f", time_used3d3.count() * 1000);
    std::chrono ::high_resolution_clock::time_point a3d4 =
        std::chrono::high_resolution_clock::now();
    checkRuntime(hipMemcpyAsync(output_array_.cpu(), output_array_.gpu(),
                                 output_array_.gpu_bytes(),
                                 hipMemcpyDeviceToHost, stream_));
    checkRuntime(hipStreamSynchronize(stream_));
    std::chrono ::high_resolution_clock::time_point a4 =
        std::chrono::high_resolution_clock::now();
    auto time_used3d4 =
        std::chrono::duration_cast<std::chrono::duration<double>>(a4 - a3d4);
//    INFO("copy output_boxarray_ time: %f", time_used3d4.count() * 1000);
    auto time_used4 =
        std::chrono::duration_cast<std::chrono::duration<double>>(a4 - a3);
    INFO("forward and decode_kernel_invoker time: %f",
         time_used4.count() * 1000);
    std::chrono ::high_resolution_clock::time_point a5 =
        std::chrono::high_resolution_clock::now();
    vector<int> arrout(num_image);
    for(int ib =0; ib < num_image ; ++ib) {
        float *parray = output_array_.cpu() +
        ib * num_class_;
        vector<float> output(num_class_);
        for(int o=0;o<num_class_;o++){
            output[o]=parray[o];
        }
        // std::cout<<ib<<"are"<<std::endl;
        // for(auto out:output){
        //     std::cout<<out<<" ";
        // }
        arrout[ib]=postprocess(output);
    }
    
    

    
    

    return arrout;
  }
};
    Infer *loadraw(const std::string &engine_file, Type type) {
    InferImpl *impl = new InferImpl();
    if (!impl->load(engine_file, type)) {
        delete impl;
        impl = nullptr;
    }
    return impl;
    }
    shared_ptr<Infer> load(const string &engine_file, Type type) {
        return std::shared_ptr<InferImpl>((InferImpl *)loadraw(
            engine_file, type));
    }
}
